#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "FrameProcessor.h"

// Optimized dimensions - divisible by 32 for warp alignment
constexpr int CROP_WIDTH = FrameProcessor::CROP_WIDTH;
constexpr int CROP_HEIGHT = FrameProcessor::CROP_HEIGHT;

__global__ void nv12ToRgbCropKernel(const uint8_t* __restrict__ nv12Data,
                                    uint8_t* __restrict__ rgbData,
                                    int srcWidth, int srcHeight,
                                    int cropX, int cropY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= CROP_WIDTH || y >= CROP_HEIGHT) return;

    // Source coordinates (with horizontal and vertical flip)
    int srcX = srcWidth - 1 - (cropX + x);   // Flip horizontally
    int srcY = srcHeight - 1 - (cropY + y);  // Flip vertically

    // Y plane
    int yIndex = srcY * srcWidth + srcX;
    int yValue = nv12Data[yIndex];

    // UV plane (interleaved, half resolution)
    int uvIndex = srcHeight * srcWidth + (srcY / 2) * srcWidth + (srcX & ~1);
    int uValue = nv12Data[uvIndex];
    int vValue = nv12Data[uvIndex + 1];

    // YUV to RGB conversion (using integer math for speed)
    int c = yValue - 16;
    int d = uValue - 128;
    int e = vValue - 128;

// Clamp macro
#define CLAMP(x) ((x) < 0 ? 0 : ((x) > 255 ? 255 : (x)))

    int r = CLAMP((298 * c + 409 * e + 128) >> 8);
    int g = CLAMP((298 * c - 100 * d - 208 * e + 128) >> 8);
    int b = CLAMP((298 * c + 516 * d + 128) >> 8);

#undef CLAMP

    // Output RGB (interleaved)
    int rgbIndex = (y * CROP_WIDTH + x) * 3;
    rgbData[rgbIndex] = b;      // Blue first
    rgbData[rgbIndex + 1] = g;  // Green second
    rgbData[rgbIndex + 2] = r;  // Red last
}

// Host function callable from C++
extern "C" void launchNv12ToRgbCrop(const uint8_t* d_nv12,
                                    uint8_t* d_rgb,
                                    int srcWidth, int srcHeight,
                                    int cropX, int cropY,
                                    hipStream_t stream) {
    dim3 blockSize(32, 16);  // 512 threads per block
    dim3 gridSize(
        (CROP_WIDTH + blockSize.x - 1) / blockSize.x,
        (CROP_HEIGHT + blockSize.y - 1) / blockSize.y);

    nv12ToRgbCropKernel<<<gridSize, blockSize, 0, stream>>>(
        d_nv12, d_rgb, srcWidth, srcHeight, cropX, cropY);
}